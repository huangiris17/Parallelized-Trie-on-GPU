#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstring>

#define MAX_NODES (1024*1024)
#define CHAR_SET_SIZE 26 // lowercase letters for now

__device__ __host__ inline int charToIndex(char c) {
    return c - 'a';
}

struct TrieNode {
    unsigned int bitmap;
    int children[CHAR_SET_SIZE];
    bool is_end_of_word;
};

class Trie{
    public:

    TrieNode* d_trie;
    int* d_node_index;
    int total_nodes;

    Trie(int max_nodes = MAX_NODES) : total_nodes(max_nodes){
        size_t num_bytes = total_nodes * sizeof(TrieNode);
        hipMalloc((void**)&d_trie, num_bytes);

        TrieNode h_root;
        h_root.bitmap = 0;
        memset(h_root.children, -1, sizeof(h_root.children));
        h_root.is_end_of_word = false;

        hipMemcpy(d_trie, &h_root, sizeof(TrieNode), hipMemcpyHostToDevice);

        int h_node_index = 1;

        hipMalloc((void**)&d_node_index, sizeof(int));
        hipMemcpy(d_node_index, &h_node_index, sizeof(int), hipMemcpyHostToDevice);
    }

    // Destructor
    ~Trie(){
        hipFree(d_trie);
        hipFree(d_node_index);
    }
};
